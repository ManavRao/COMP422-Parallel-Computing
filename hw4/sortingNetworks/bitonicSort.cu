#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */



//Based on http://www.iti.fh-flensburg.de/lang/algorithmen/sortieren/bitonic/bitonicen.htm



#include <assert.h>
#include <hip/hip_runtime_api.h>
#include "sortingNetworks_common.h"
#include "sortingNetworks_common.cuh"

#define DEVICE_MEM 1048576


////////////////////////////////////////////////////////////////////////////////
// Monolithic bitonic sort kernel for short arrays fitting into shared memory
////////////////////////////////////////////////////////////////////////////////
__global__ void bitonicSortShared(
        uint *d_DstKey,
        uint *d_SrcKey,
        uint arrayLength,
        uint dir
        )
{
    //Shared memory storage for one or more short vectors
    __shared__ uint s_key[SHARED_SIZE_LIMIT];

    //Offset to the beginning of subbatch and load data
    d_SrcKey += blockIdx.x * SHARED_SIZE_LIMIT + threadIdx.x;
    d_DstKey += blockIdx.x * SHARED_SIZE_LIMIT + threadIdx.x;
    s_key[threadIdx.x +                       0] = d_SrcKey[                      0];
    s_key[threadIdx.x + (SHARED_SIZE_LIMIT / 2)] = d_SrcKey[(SHARED_SIZE_LIMIT / 2)];

    for (uint size = 2; size < arrayLength; size <<= 1)
    {
        //Bitonic merge
        uint ddd = dir ^ ((threadIdx.x & (size / 2)) != 0);

        for (uint stride = size / 2; stride > 0; stride >>= 1)
        {
            __syncthreads();
            uint pos = 2 * threadIdx.x - (threadIdx.x & (stride - 1));
            Comparator(
                    s_key[pos +      0], 
                    s_key[pos + stride],
                    ddd
                    );
        }
    }

    //ddd == dir for the last bitonic merge step
    {
        for (uint stride = arrayLength / 2; stride > 0; stride >>= 1)
        {
            __syncthreads();
            uint pos = 2 * threadIdx.x - (threadIdx.x & (stride - 1));
            Comparator(
                    s_key[pos +      0],
                    s_key[pos + stride],
                    dir
                    );
        }
    }

    __syncthreads();
    d_DstKey[                      0] = s_key[threadIdx.x +                       0];
    d_DstKey[(SHARED_SIZE_LIMIT / 2)] = s_key[threadIdx.x + (SHARED_SIZE_LIMIT / 2)];
}



////////////////////////////////////////////////////////////////////////////////
// Bitonic sort kernel for large arrays (not fitting into shared memory)
////////////////////////////////////////////////////////////////////////////////
//Bottom-level bitonic sort
//Almost the same as bitonicSortShared with the exception of
//even / odd subarrays being sorted in opposite directions
//Bitonic merge accepts both
//Ascending | descending or descending | ascending sorted pairs
__global__ void bitonicSortShared1(
        uint *d_DstKey,
        uint *d_SrcKey
        )
{
    //Shared memory storage for current subarray
    __shared__ uint s_key[SHARED_SIZE_LIMIT];

    //Offset to the beginning of subarray and load data
    d_SrcKey += blockIdx.x * SHARED_SIZE_LIMIT + threadIdx.x;
    d_DstKey += blockIdx.x * SHARED_SIZE_LIMIT + threadIdx.x;
    s_key[threadIdx.x +                       0] = d_SrcKey[                      0];
    s_key[threadIdx.x + (SHARED_SIZE_LIMIT / 2)] = d_SrcKey[(SHARED_SIZE_LIMIT / 2)];

    for (uint size = 2; size < SHARED_SIZE_LIMIT; size <<= 1)
    {
        //Bitonic merge
        uint ddd = (threadIdx.x & (size / 2)) != 0;

        for (uint stride = size / 2; stride > 0; stride >>= 1)
        {
            __syncthreads();
            uint pos = 2 * threadIdx.x - (threadIdx.x & (stride - 1));
            Comparator(
                    s_key[pos +      0],
                    s_key[pos + stride],
                    ddd
                    );
        }
    }

    //Odd / even arrays of SHARED_SIZE_LIMIT elements
    //sorted in opposite directions
    uint ddd = blockIdx.x & 1;
    {
        for (uint stride = SHARED_SIZE_LIMIT / 2; stride > 0; stride >>= 1)
        {
            __syncthreads();
            uint pos = 2 * threadIdx.x - (threadIdx.x & (stride - 1));
            Comparator(
                    s_key[pos +      0],
                    s_key[pos + stride],
                    ddd
                    );
        }
    }


    __syncthreads();
    d_DstKey[                      0] = s_key[threadIdx.x +                       0];
    d_DstKey[(SHARED_SIZE_LIMIT / 2)] = s_key[threadIdx.x + (SHARED_SIZE_LIMIT / 2)];
}

//Bitonic merge iteration for stride >= SHARED_SIZE_LIMIT
__global__ void bitonicMergeGlobal(
        uint *d_DstKey,
        uint *d_SrcKey,
        uint arrayLength,
        uint size,
        uint stride,
        uint dir
        )
{
    uint global_comparatorI = blockIdx.x * blockDim.x + threadIdx.x;
    uint        comparatorI = global_comparatorI & (arrayLength / 2 - 1);

    //Bitonic merge
    uint ddd = dir ^ ((comparatorI & (size / 2)) != 0);
    uint pos = 2 * global_comparatorI - (global_comparatorI & (stride - 1));

    uint keyA = d_SrcKey[pos +      0];
    uint keyB = d_SrcKey[pos + stride];

    Comparator(
            keyA,
            keyB,
            ddd
            );

    d_DstKey[pos +      0] = keyA;
    d_DstKey[pos + stride] = keyB;
}

//Combined bitonic merge steps for
//size > SHARED_SIZE_LIMIT and stride = [1 .. SHARED_SIZE_LIMIT / 2]
__global__ void bitonicMergeShared(
        uint *d_DstKey,
        uint *d_SrcKey,
        uint arrayLength,
        uint size,
        uint dir
        )
{
    //Shared memory storage for current subarray
    __shared__ uint s_key[SHARED_SIZE_LIMIT];

    d_SrcKey += blockIdx.x * SHARED_SIZE_LIMIT + threadIdx.x;
    d_DstKey += blockIdx.x * SHARED_SIZE_LIMIT + threadIdx.x;
    s_key[threadIdx.x +                       0] = d_SrcKey[                      0];
    s_key[threadIdx.x + (SHARED_SIZE_LIMIT / 2)] = d_SrcKey[(SHARED_SIZE_LIMIT / 2)];

    //Bitonic merge
    uint comparatorI = UMAD(blockIdx.x, blockDim.x, threadIdx.x) & ((arrayLength / 2) - 1);
    uint ddd = dir ^ ((comparatorI & (size / 2)) != 0);

    for (uint stride = SHARED_SIZE_LIMIT / 2; stride > 0; stride >>= 1)
    {
        __syncthreads();
        uint pos = 2 * threadIdx.x - (threadIdx.x & (stride - 1));
        Comparator(
                s_key[pos +      0],
                s_key[pos + stride],
                ddd
                );
    }

    __syncthreads();
    d_DstKey[                      0] = s_key[threadIdx.x +                       0];
    d_DstKey[(SHARED_SIZE_LIMIT / 2)] = s_key[threadIdx.x + (SHARED_SIZE_LIMIT / 2)];
}



////////////////////////////////////////////////////////////////////////////////
// Interface function
////////////////////////////////////////////////////////////////////////////////
//Helper function (also used by odd-even merge sort)
extern "C" uint factorRadix2(uint *log2L, uint L)
{
    if (!L)
    {
        *log2L = 0;
        return 0;
    }
    else
    {
        for (*log2L = 0; (L & 1) == 0; L >>= 1, *log2L++);

        return L;
    }
}

extern "C" uint bitonicSort(
        uint *d_DstKey,
        uint *d_SrcKey,
        uint *h_SrcKey,
        uint arrayLength,
        uint dir
        )
{
    //Nothing to sort
    if (arrayLength < 2)
        return 0;

    //Only power-of-two array lengths are supported by this implementation
    uint log2L;
    uint factorizationRemainder = factorRadix2(&log2L, arrayLength);
    assert(factorizationRemainder == 1);

    dir = (dir != 0);

    uint blockCount, threadCount;
    uint flag;
    hipError_t err;
    if(arrayLength > DEVICE_MEM){
        blockCount = DEVICE_MEM / SHARED_SIZE_LIMIT;
        threadCount = SHARED_SIZE_LIMIT / 2;
        flag = 1;
    }
    else if (arrayLength <= DEVICE_MEM && arrayLength > SHARED_SIZE_LIMIT){
        blockCount = arrayLength  / SHARED_SIZE_LIMIT;
        threadCount = SHARED_SIZE_LIMIT / 2;
        flag = 2;
    }

    else{
        blockCount = 1;
        threadCount = SHARED_SIZE_LIMIT / 2;
        flag = 0;
    }

    if (flag == 0){
        err = hipMemcpy(d_SrcKey, h_SrcKey, arrayLength * sizeof(uint), hipMemcpyHostToDevice);
        checkCudaErrors(err);
        bitonicSortShared<<<blockCount, threadCount>>>(d_DstKey, d_SrcKey, arrayLength, dir);
        err = hipMemcpy(h_SrcKey, d_DstKey, arrayLength * sizeof(uint), hipMemcpyDeviceToHost);
        checkCudaErrors(err);
    }
    else
    {
        for(uint i = 0; i * DEVICE_MEM < arrayLength ; i++){
            uint copy_size = (flag == 1) ? DEVICE_MEM : arrayLength;  
            err = hipMemcpy(d_SrcKey, h_SrcKey + i * DEVICE_MEM, copy_size * sizeof(uint), hipMemcpyHostToDevice);
            checkCudaErrors(err);
            bitonicSortShared1<<<blockCount, threadCount>>>(d_DstKey, d_SrcKey);
            err = hipMemcpy(h_SrcKey + i * DEVICE_MEM, d_DstKey, copy_size * sizeof(uint), hipMemcpyDeviceToHost);
            checkCudaErrors(err);
        }
        uint start_size = (flag == 2) ? arrayLength : DEVICE_MEM;
        for(uint size = start_size; size <= arrayLength; size <<= 1){
            for (unsigned stride = size / 2; stride > 0; stride >>= 1){
                if (stride >= DEVICE_MEM){ // Comparing/Merging discontinous subarrays seperated by a distance greater than 2^20
                    for (uint i = 0; i < arrayLength / DEVICE_MEM; i++){
                        err = hipMemcpy(d_SrcKey, h_SrcKey + i*DEVICE_MEM/2 + (i*(DEVICE_MEM/2)/stride)*stride, DEVICE_MEM/2*sizeof(uint), hipMemcpyHostToDevice);
                        checkCudaErrors(err);
                        err = hipMemcpy(d_SrcKey + DEVICE_MEM/2, h_SrcKey + i*DEVICE_MEM/2 + (i*(DEVICE_MEM/2)/stride+1)*stride, DEVICE_MEM/2*sizeof(uint), hipMemcpyHostToDevice);
                        uint ddd = dir ^ ((i*DEVICE_MEM/size) & 1);
                        bitonicMergeGlobal<<<(blockCount * threadCount) /256, 256>>>(d_DstKey, d_SrcKey, arrayLength, size, DEVICE_MEM/2, ddd);
                        err = hipMemcpy(h_SrcKey + i*DEVICE_MEM/2 + (i*(DEVICE_MEM/2)/stride)*stride, d_DstKey, DEVICE_MEM/2 * sizeof(uint), hipMemcpyDeviceToHost);
                        checkCudaErrors(err);
                        err = hipMemcpy(h_SrcKey + i*DEVICE_MEM/2 + (i*(DEVICE_MEM/2)/stride+1)*stride, d_DstKey+DEVICE_MEM/2, DEVICE_MEM/2 * sizeof(uint), hipMemcpyDeviceToHost);
                        checkCudaErrors(err);

                    }
                }
                else{
                    for(uint i = 0; i * DEVICE_MEM < arrayLength ; i++){ //Moving DEVICE_MEM to the left side of the condition so that the first iteration is gauranteed to be executed
                        err = hipMemcpy(d_DstKey, h_SrcKey + i*DEVICE_MEM, start_size * sizeof(uint), hipMemcpyHostToDevice);
                        checkCudaErrors(err);
                        uint ddd = dir ^ ((i*DEVICE_MEM/size) & 1);
                        uint start = 2 * stride;
                        if(size == start_size){
                            start = 2 * SHARED_SIZE_LIMIT; // for the 1st iteration, the size of bitonic sequence is 2048
                        }
                        for(uint sz = start; sz <= start_size; sz <<= 1){ 
                            for(unsigned sd = sz / 2; sd > 0; sd >>=1){
                                if(sd >= SHARED_SIZE_LIMIT){
                                    bitonicMergeGlobal<<<(blockCount*threadCount)/ 256,256>>>(d_DstKey, d_DstKey, arrayLength, sz, sd, ddd);
                                }
                                else{
                                    bitonicMergeShared<<<blockCount, threadCount>>>(d_DstKey, d_DstKey, arrayLength, sz, ddd);
                                    break;
                                }
                            }
                        }
                        err = hipMemcpy(h_SrcKey + i*DEVICE_MEM, d_DstKey, start_size*sizeof(uint), hipMemcpyDeviceToHost);
                        checkCudaErrors(err);
                    }
                    break;
                }
            }
        }
    }

    return threadCount; 
}
